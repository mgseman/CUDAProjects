
#include "hip/hip_runtime.h"


#include <stdio.h>

void addWithCuda(float *c, float *a, float *b, unsigned int size);

__global__ void addKernel(float* c, float* a, float* b)
{
    // threadIdx is the local thread number 
    // blockIdx is the block number that contains the local threads
    // blockDim specifies the total number of threads in each block
    // threadIdx, blockIdx and blockDim each have multiple dimensions x, y and z
    // Use appropriate dimensions for dimension of data being used

    // Unique global index for GPU threads
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Pair wise addition
    c[i] = a[i] + b[i];
}

int main()
{
    const int n = 5;
    float h_a[n] = { 1.1, 2.2, 3.3, 4.4, 5.5 };
    float h_b[n] = { 10, 20, 30, 40, 50 };
    float h_c[n] = { 0 };

    // Add vectors in parallel.
    addWithCuda(h_c, h_a, h_b, n);

    printf("{1.1,2.2,3.3,4.4,5.5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        h_c[0], h_c[1], h_c[2], h_c[3], h_c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipDeviceReset();

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(float *c, float *a, float *b, unsigned int n)
{
    // Aloccate device variables
    float *d_a = 0;
    float *d_b = 0;
    float *d_c = 0;

    int n_blocks = 1;
    int n_thds = n;

    // Size of vectors in bytes for cudamalloc
    int size = n * sizeof(float);

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    // Allocate GPU buffers for three vectors (two input, one output)    .
    hipMalloc((void**)&d_c, size);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    // <<< number of thread blocks, number of threads in each block >>>
    addKernel<<<n_blocks, n_thds>>>(d_c, d_a, d_b);

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_c);
    hipFree(d_a);
    hipFree(d_b);
}
